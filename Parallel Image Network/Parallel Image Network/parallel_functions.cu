#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <>
#include "image_properties.h"
#include "array_properties.h"
#include "shared_functions.h"
#include "serial_functions.h"

__global__ void
parallel_train_network(image_chunk * image_grid, network_chunk * network_grid)
{
	char bit_value = image_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) +
		(blockIdx.z*VER_ARRAYS*HOR_ARRAYS)].image_data[threadIdx.x%64][threadIdx.x/64];
	/*In this case, where there are necessarily more variables than can be included in a single launch
	due to needing to specify more dimensions than can be arranged in blocks (limited to three) 
	and adding thread dimensions would exceed the hardware limit (512^2 = 262,144, 
	even the best GPU can't exceed 2048) it's a case of choosing between multiple kernel launches, 
	or including a loop in the device code. In this case, I chose to have just one loop here.*/
	for (int i = 0; i < blockDim.x; i++)
	{
		char relating_bit_value = image_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) + 
			(blockIdx.z*VER_ARRAYS*HOR_ARRAYS)].image_data[i % 64][i / 64];
		if (relating_bit_value == bit_value)
		{
			network_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) +
				(blockIdx.z*VER_ARRAYS*HOR_ARRAYS)].network_weights[threadIdx.x][i] += 1;
		}
		else
		{
			network_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) +
				(blockIdx.z*VER_ARRAYS*HOR_ARRAYS)].network_weights[threadIdx.x][i] += -1;
		}
	}
}

__global__ void
parallel_recall_image(image_chunk * image_grid, network_chunk * network_grid, int colour_channel) {
	//accessible to all threads within a block.
	__shared__ int neuron_output[BITS_PER_SUBIMAGE];
	/*Unlike the above code, here there are separate kernels for each colour channel--otherwise there
	would be nested for loops in the device code. Because of the need to access a shared array, this
	has the potential for leaving lots of threads waiting most of the time, which should be avoided
	as it's poor performance.*/
	char weighted_value = image_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) +
		(colour_channel*VER_ARRAYS*HOR_ARRAYS)].image_data[threadIdx.x % 64][threadIdx.x / 64] * 2 - 1; 
	neuron_output[threadIdx.x] = network_grid[blockIdx.x + blockIdx.y*VER_ARRAYS +
		colour_channel*VER_ARRAYS*HOR_ARRAYS].network_weights[blockIdx.z][threadIdx.x] * weighted_value;

	__syncthreads();

	for (int i = blockDim.x / 2; i > 0; i >>= 1) { //first half of the array adds the second
		//then first half of THAT adds the second, until everything has been added to the 0th element
		if (threadIdx.x < i) {
			neuron_output[threadIdx.x] += neuron_output[threadIdx.x + i];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) //only operate this on one thread
		//sets bit values
	{
		if (neuron_output[0] > 0) { 
			image_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) + (colour_channel*VER_ARRAYS*HOR_ARRAYS)]
				.image_data[blockIdx.z % 64][blockIdx.z / 64] = 1;
		}
		else {
			image_grid[blockIdx.x + (blockIdx.y*VER_ARRAYS) + (colour_channel*VER_ARRAYS*HOR_ARRAYS)]
				.image_data[blockIdx.z % 64][blockIdx.z / 64] = 0;
		}
	}
}

__host__
void
parallel_trainer(image_chunk ***image_grid, network_chunk ***network_grid) {

	//convert the multidimensional array  to a single array--can be transferred in one go to the GPU
	//rather than specifying the dimensions in the method header.
	//as we can see here, using the Cuda version 6.0+ unified memory allows for transferring structs
	//rather than having to allocate memory for each sub-array individually and update the device pointers
	image_chunk * parallel_image;
	hipMallocManaged(&parallel_image, sizeof(image_chunk)*VER_ARRAYS*HOR_ARRAYS*COLOUR_CHANNELS);
	network_chunk * parallel_network;
	hipMallocManaged(&parallel_network, sizeof(network_chunk)*VER_ARRAYS*HOR_ARRAYS*COLOUR_CHANNELS);
	for (int chunk_x = 0; chunk_x < VER_ARRAYS; chunk_x++)
	{
		for (int chunk_y = 0; chunk_y < HOR_ARRAYS; chunk_y++)
		{
			for (int colour = 0; colour < COLOUR_CHANNELS; colour++)
			{
				for (int image_y = 0; image_y < 8; image_y++)
				{
					for (int image_x = 0; image_x < 64; image_x++)
					{
						parallel_image[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.image_data[image_x][image_y]
							= image_grid[chunk_x][chunk_y][colour].image_data[image_x][image_y];
					}
				}
				for (int relationship_x = 0; relationship_x < BITS_PER_SUBIMAGE; relationship_x++)
				{
					for (int relationship_y = 0; relationship_y < BITS_PER_SUBIMAGE; relationship_y++)
					{
						parallel_network[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.network_weights[relationship_x][relationship_y]
							= network_grid[chunk_x][chunk_y][colour].network_weights[relationship_x][relationship_y];
					}
				}
			}
		}
	}	

	dim3 grid_dimensions(VER_ARRAYS, HOR_ARRAYS, COLOUR_CHANNELS); //dimensions of the block grid
	//e.g., how many blocks to have
	dim3 block_dimensions(BITS_PER_SUBIMAGE); //number of threads per block. With one value, the Y and Z dimensions
	//are initialised to 1.

	parallel_train_network<<<grid_dimensions, block_dimensions>>>(parallel_image, parallel_network);
	hipDeviceSynchronize();

	//copy back over to the original arrays
	for (int chunk_x = 0; chunk_x < VER_ARRAYS; chunk_x++)
	{
		for (int chunk_y = 0; chunk_y < HOR_ARRAYS; chunk_y++)
		{
			for (int colour = 0; colour < COLOUR_CHANNELS; colour++)
			{
				for (int image_y = 0; image_y < 8; image_y++)
				{
					for (int image_x = 0; image_x < 64; image_x++)
					{
						image_grid[chunk_x][chunk_y][colour].image_data[image_x][image_y] =
							parallel_image[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.image_data[image_x][image_y];
					}
				}
				for (int relationship_x = 0; relationship_x < BITS_PER_SUBIMAGE; relationship_x++)
				{
					for (int relationship_y = 0; relationship_y < BITS_PER_SUBIMAGE; relationship_y++)
					{
						network_grid[chunk_x][chunk_y][colour].network_weights[relationship_x][relationship_y] =
						parallel_network[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.network_weights[relationship_x][relationship_y];
					}
				}
			}
		}
	}

	hipFree(parallel_image);
	hipFree(parallel_network);

}

__host__
void
parallel_recall(image_chunk ***image_grid, network_chunk ***network_grid, int loops) {
	//as above, copy the data into linear arrays
	image_chunk * parallel_image;
	hipMallocManaged(&parallel_image, sizeof(image_chunk)*VER_ARRAYS*HOR_ARRAYS*COLOUR_CHANNELS);
	network_chunk * parallel_network;
	hipMallocManaged(&parallel_network, sizeof(network_chunk)*VER_ARRAYS*HOR_ARRAYS*COLOUR_CHANNELS);
	for (int chunk_x = 0; chunk_x < VER_ARRAYS; chunk_x++)
	{
		for (int chunk_y = 0; chunk_y < HOR_ARRAYS; chunk_y++)
		{
			for (int colour = 0; colour < COLOUR_CHANNELS; colour++)
			{
				for (int image_y = 0; image_y < 8; image_y++)
				{
					for (int image_x = 0; image_x < 64; image_x++)
					{
						parallel_image[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.image_data[image_x][image_y]
							= image_grid[chunk_x][chunk_y][colour].image_data[image_x][image_y];
					}
				}
				for (int relationship_x = 0; relationship_x < BITS_PER_SUBIMAGE; relationship_x++)
				{
					for (int relationship_y = 0; relationship_y < BITS_PER_SUBIMAGE; relationship_y++)
					{
						parallel_network[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.network_weights[relationship_x][relationship_y]
							= network_grid[chunk_x][chunk_y][colour].network_weights[relationship_x][relationship_y];
					}
				}
			}
		}
	}

	dim3 grid_dimensions(VER_ARRAYS, HOR_ARRAYS, BITS_PER_SUBIMAGE);
	dim3 block_dimensions(BITS_PER_SUBIMAGE); 
	for (int i = 0; i < loops; i++)
	{
		for (int colour = 0; colour < COLOUR_CHANNELS; colour++)
		{
			parallel_recall_image<<<grid_dimensions, block_dimensions>>>(parallel_image, parallel_network, colour);
			hipDeviceSynchronize(); //makes sure that the program doesn't proceed until the device code is finished executing
			//all data is already on the device, so it's just a matter of launching another kernel
		}
	}
	//copy back over to the original arrays
	for (int chunk_x = 0; chunk_x < VER_ARRAYS; chunk_x++)
	{
		for (int chunk_y = 0; chunk_y < HOR_ARRAYS; chunk_y++)
		{
			for (int colour = 0; colour < COLOUR_CHANNELS; colour++)
			{
				for (int image_y = 0; image_y < 8; image_y++)
				{
					for (int image_x = 0; image_x < 64; image_x++)
					{
						image_grid[chunk_x][chunk_y][colour].image_data[image_x][image_y] =
							parallel_image[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.image_data[image_x][image_y];
					}
				}
				for (int relationship_x = 0; relationship_x < BITS_PER_SUBIMAGE; relationship_x++)
				{
					for (int relationship_y = 0; relationship_y < BITS_PER_SUBIMAGE; relationship_y++)
					{
						network_grid[chunk_x][chunk_y][colour].network_weights[relationship_x][relationship_y] =
							parallel_network[chunk_x + (chunk_y * VER_ARRAYS) + (colour * VER_ARRAYS * HOR_ARRAYS)]
							.network_weights[relationship_x][relationship_y];
					}
				}
			}
		}
	}

	hipFree(parallel_image);
	hipFree(parallel_network);

}